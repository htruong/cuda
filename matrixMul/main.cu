#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <assert.h>


// CUDA runtime
#include <hip/hip_runtime.h>

typedef unsigned int uint;

using namespace std;

// C[m,k] = A[m,n] * B[n,k]
// m rows, k columns

// C[m,k] is stored as such
// i[m=1, k=1], i[m=1, k=2],
// i[m=2, k=1], i[m=2, k=2], and so on


void print_matrix(float *matrix, uint size) {
  uint how_many_elements = 25;

  if (size < 25) {
    how_many_elements = size;
  }

  for (uint i = 0; i < how_many_elements; i++) {
    printf(" %f", matrix[i]);
  }

  printf("\n");
}

void host_matmul(float *a, float *b, float *c, uint m, uint n, uint k) {
  // Go row by row
  for (uint i = 0; i < m; i++) {
    // Go column by column
    for (uint j = 0; j < k; j++) {
      // Go through the cells of each dest matrix
      for (uint t = 0; t < n; t++) {
	// c (i, j) += a(i, t) * b(t, j)
	c[i * k + j] += a[i * n + t] * b[t * k + j];
      }
    }
  }
}

__global__ void kernel_mathmul(float * a, float * b, float * c, uint k, uint n)
{
	// This does not work yet when the columns count of the first matrix is > 256 or whatever...
    uint i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < k) {
      /*
      printf("[%d] ", i);

	  if (i == 0) {
		  printf("a now is: ");
	  	  for (uint j = 0; j< n; j++) {
		  	  printf("%f ", a[j]);
	  	  }
	  	  printf("\n");
	  }
	  */
      c[i] = 0; // reset it first, remember it doesn't get cleared everytime
      
      for (uint t=0; t<n; t++) {
    	  	  c[i] += a[t] * b[i + t*k];
      }
    }

}

void dev_matmul(float *a, float *b, float *c, uint m, uint n, uint k) {
    int threads_per_block = 256;
    int blocks =  ceil((float)(m) / threads_per_block);
    // printf("Number of blocks: [%d]\n", blocks);
    
    float * dev_a_onerow;
	hipMalloc(&dev_a_onerow, n*sizeof(float));
    
    float * dev_c_onerow;
    hipMalloc(&dev_c_onerow, k*sizeof(float));
    
    float * dev_b;
    hipMalloc(&dev_b, m*n*sizeof(float));
    
    hipMemcpy(dev_b, b, m*n*sizeof(float), hipMemcpyHostToDevice);
    
    // Compute the rows of resulting matrix one line by one line.
    for (uint i = 0; i < m; i++) {
      // copy one row of the a matrix to the device
      hipMemcpy(dev_a_onerow, a + (i * n), n * sizeof(float), hipMemcpyHostToDevice);

      //printf("Calling kernel mathmul\n");
      kernel_mathmul <<<blocks,threads_per_block>>> (dev_a_onerow, dev_b, dev_c_onerow, k, n);
      hipDeviceSynchronize();

      // copy the resulting row back
      hipMemcpy(c + i * k, dev_c_onerow, k * sizeof(float), hipMemcpyDeviceToHost);

      //print_matrix(c, m*k);
    }
    
    hipFree(dev_a_onerow);
    hipFree(dev_b);
    hipFree(dev_c_onerow);
}

void init_matrix(float *matrix, uint size) {
  for (uint i = 0; i < size; i++) {
    matrix[i] = ((float) rand()) / RAND_MAX;
    //matrix[i] = 1.0;
  }
}

void clear_matrix(float *matrix, uint size) {
  for (uint i = 0; i < size; i++) {
    matrix[i] = 0.0;
  }
}


void verify_matrix(const float *matrix1, const float *matrix2, const uint size) {
  for (uint i = 0; i < size; i++) {
    assert(matrix1[i] == matrix2[i]);
  }
}



int main() {
  uint m = 200;
  uint n = 200;
  uint k = 200;
  
  srand( time (NULL) );
  
  float * a_matrix = new float[m*n];
  float * b_matrix = new float[n*k];
  float * c_matrix = new float[m*k];
  float * c_dev_matrix = new float[m*k];

  clock_t start;
  clock_t end;
  /*
  printf("Initializing matrices with sample numbers\n", 2, 2);
  a_matrix[0] = 1; a_matrix[2] = -2; a_matrix[4] = 3; a_matrix[5] = -1;
  b_matrix[1] = 3; b_matrix[2] = -2; b_matrix[3] = -1; b_matrix[5] = 4;
  
  printf("Here comes the first 25 elements of a, b, c before:\n");
  print_matrix(a_matrix, m*n);
  print_matrix(b_matrix, n*k);
  print_matrix(c_matrix, m*k);
  
  printf("Do host-calculation:\n");
  host_matmul(a_matrix, b_matrix, c_matrix, m, n, k);
  
  printf("Here comes the first 25 elements of a, b, c after:\n");
  print_matrix(a_matrix, m*n);
  print_matrix(b_matrix, n*k);
  print_matrix(c_matrix, m*k);
  
  printf("Asserting that host-calculation is correct: ");
  assert(c_matrix[0] == 0);
  assert(c_matrix[1] == -5);
  assert(c_matrix[2] == -6);
  assert(c_matrix[3] == -7);
  printf("It is.\n");
  ////////////////////////////////////////////////
  */
  printf("Intialize random matrices: ");
  init_matrix(a_matrix, m*n);
  init_matrix(b_matrix, n*k);
  printf("Done. \n");
  print_matrix(a_matrix, m*n);
  print_matrix(b_matrix, n*k);
  print_matrix(c_matrix, m*k);
  ///////////////////////////////////////////////

  printf("Do host-calculation: ");
  clear_matrix(c_matrix, m*k);
  start = clock();
  host_matmul(a_matrix, b_matrix, c_matrix, m, n, k);
  end = clock();
  printf("Took %d clock cycles\n", end - start);

  printf("Here comes the first 25 elements of c:\n");
  print_matrix(c_matrix, m*k);

  printf("Do device-calculation: ");
  start = clock();
  dev_matmul(a_matrix, b_matrix, c_dev_matrix, m, n, k);
  end = clock();
  printf("Took %d clock cycles\n", end - start);

  printf("Here comes the first 25 elements of c on device:\n");
  print_matrix(c_dev_matrix, m*k);
  printf("Checking if GPU result is correct: ");
  verify_matrix(c_matrix, c_dev_matrix, m*k);
  printf("It is.");

  delete a_matrix;
  delete b_matrix;
  delete c_matrix;
  delete c_dev_matrix;
}
