#define LIMIT -999
#define BLOCK_SIZE 16

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "needle_gpu.h"

// includes, kernels
#include "needle_kernel_diagonal.cu"

/* Private structure, do not bother */
struct needle_context {
	unsigned int gpu_num;
	char *sequence_set1;
	char *sequence_set2;
	unsigned int *pos1;
	unsigned int *pos2;
	int *score_matrix;
	unsigned int *pos_matrix;
	unsigned int max_pair_no;
	short penalty;
	// Grunt work... eww
	char *d_sequence_set1_h1, *d_sequence_set2_h1, *d_sequence_set1_h2, *d_sequence_set2_h2;
	unsigned int *d_pos1_h1, *d_pos2_h1, *d_pos_matrix_h1, *d_pos1_h2, *d_pos2_h2, *d_pos_matrix_h2;
	int *d_score_matrix_h1, *d_score_matrix_h2;
	hipStream_t *stream1, *stream2;
	unsigned int half_b, other_half_b, max_length_per_seq;
};

double gettime(){
	struct timeval t;
	gettimeofday(&t,NULL);
	return t.tv_sec+t.tv_usec*1e-6;
}


void memcpy_and_run (
				unsigned int begin,
				unsigned int end,
				hipStream_t * stream,
				char *sequence_set1,
				char *sequence_set2,
				char *d_sequence_set1,
				char *d_sequence_set2,
				unsigned int *pos1,
				unsigned int *pos2,
				unsigned int *d_pos1,
				unsigned int *d_pos2,
				int *score_matrix,
				unsigned int *pos_matrix,
				int *d_score_matrix,
				unsigned int *d_pos_matrix,
				short penalty) {
				
		unsigned int batch_size = end-begin;
		// Memcpy to device

		#ifdef VERBOSE
		double start_marker = 0;
		start_marker = gettime();
		printf("-- Start calculation from %d to %d --\n", begin, end);
		#endif

		#ifdef DUAL_BUFFERING
		hipMemcpyAsync( d_sequence_set1, sequence_set1 + pos1[begin], sizeof(char)*(pos1[end] - pos1[begin]), hipMemcpyHostToDevice, *stream);
		hipMemcpyAsync( d_sequence_set2, sequence_set2 + pos2[begin], sizeof(char)*(pos2[end] - pos2[begin]), hipMemcpyHostToDevice, *stream);
		hipMemcpyAsync( d_pos1, pos1 /*+ begin*/, sizeof(unsigned int)*(batch_size+1), hipMemcpyHostToDevice, *stream );
		hipMemcpyAsync( d_pos2, pos2 /*+ begin*/, sizeof(unsigned int)*(batch_size+1), hipMemcpyHostToDevice, *stream );
		hipMemcpyAsync( d_pos_matrix, pos_matrix /*+ begin*/, sizeof(unsigned int)*(batch_size+1), hipMemcpyHostToDevice, *stream );
		#else
		hipMemcpy( d_sequence_set1, sequence_set1 + pos1[begin], sizeof(char)*(pos1[end] - pos1[begin]), hipMemcpyHostToDevice );
		hipMemcpy( d_sequence_set2, sequence_set2 + pos2[begin], sizeof(char)*(pos2[end] - pos2[begin]), hipMemcpyHostToDevice );
		hipMemcpy( d_pos1, pos1 /*+ begin*/, sizeof(unsigned int)*(batch_size+1), hipMemcpyHostToDevice );
		hipMemcpy( d_pos2, pos2 /*+ begin*/, sizeof(unsigned int)*(batch_size+1), hipMemcpyHostToDevice );
		hipMemcpy( d_pos_matrix, pos_matrix /*+ begin*/, sizeof(unsigned int)*(batch_size+1), hipMemcpyHostToDevice );
		#endif

		#ifdef VERBOSE
		printf("\t [%d - %d] Memcpy CPU-GPU: %f\n", begin, end, gettime() - start_marker);
		start_marker = gettime();
		#endif
		
		#ifdef DUAL_BUFFERING
		needleman_cuda_diagonal<<<batch_size,512, 0, *stream>>>(d_sequence_set1, d_sequence_set2,
				d_pos1, d_pos2,
				d_score_matrix, d_pos_matrix,
				batch_size, penalty);
		#else
		needleman_cuda_diagonal<<<batch_size,512>>>(d_sequence_set1, d_sequence_set2,
				d_pos1, d_pos2,
				d_score_matrix, d_pos_matrix,
				batch_size, penalty);
		#endif
		
		__LINE__, hipDeviceSynchronize();
		
		#ifdef VERBOSE
		printf("\t [%d - %d] Kernel: %f\n", begin, end, gettime() - start_marker);
		start_marker = gettime();
		#endif
		
		#ifdef DUAL_BUFFERING
		hipMemcpyAsync( score_matrix + pos_matrix[begin], d_score_matrix, sizeof(int)*(pos_matrix[end] - pos_matrix[begin]), hipMemcpyDeviceToHost, *stream );
		#else
		hipMemcpy( score_matrix + pos_matrix[begin], d_score_matrix, sizeof(int)*(pos_matrix[end] - pos_matrix[begin]), hipMemcpyDeviceToHost );
		#endif
		
		#ifdef VERBOSE
		printf("\t [%d - %d] Memcpy GPU-CPU: %f\n", begin, end, gettime() - start_marker);
		#endif
}

void needleman_gpu(char *sequence_set1,
				char *sequence_set2,
				unsigned int *pos1,
				unsigned int *pos2,
				int *score_matrix,
				unsigned int *pos_matrix,
				unsigned int max_pair_no,
				short penalty,
				char *d_sequence_set1_h1,
				char *d_sequence_set2_h1,
				char *d_sequence_set1_h2,
				char *d_sequence_set2_h2,
				unsigned int *d_pos1_h1,
				unsigned int *d_pos2_h1,
				unsigned int *d_pos_matrix_h1,
				unsigned int *d_pos1_h2,
				unsigned int *d_pos2_h2,
				unsigned int *d_pos_matrix_h2,
				int *d_score_matrix_h1,
				int *d_score_matrix_h2,
				hipStream_t * stream1,
				hipStream_t * stream2,
				needle_context * ctx
				)
{
	bool done = false;

	unsigned int start = 0;
	unsigned int end = 0;
	bool turn = true;
	while (!done) {
		int tmp_batch_sz = turn ? ctx->half_b : ctx->other_half_b;
		if (start + tmp_batch_sz > max_pair_no) {
			end = max_pair_no;
			done = true;
		} else {
			end = start + tmp_batch_sz;
		}
		
		memcpy_and_run (
			start,
			end,
			turn ? stream1 : stream2 ,
			sequence_set1,
			sequence_set2,
			turn ? d_sequence_set1_h1 : d_sequence_set1_h2,
			turn ? d_sequence_set2_h1 : d_sequence_set2_h2,
			pos1,
			pos2,
			turn ? d_pos1_h1 : d_pos1_h2,
			turn ? d_pos2_h1 : d_pos2_h2,
			score_matrix,
			pos_matrix,
			turn ? d_score_matrix_h1 : d_score_matrix_h2,
			turn ? d_pos_matrix_h1 : d_pos_matrix_h2,
			penalty);
				
		start = end;
		#ifdef DUAL_BUFFERING
		turn = !turn;
		#endif
	}
	hipDeviceSynchronize();
	
}

void * needle_init(
	const int gpu_num,
	unsigned int max_length_per_seq,
	char *sequence_set1,
	char *sequence_set2,
	unsigned int *pos1,
	unsigned int *pos2,
	int *score_matrix, 
	unsigned int *pos_matrix
	)
{
	printf("-- NEEDLEMAN MODULE INITIALIZING --\n", 0);
	
	hipSetDevice(gpu_num);
	
	double start_marker; // Start time marker

	// First we need to see how to devide the memory...
	// Query the device capabilities to see how much we can allocate for this problem

	size_t freeMem = 0;
	size_t totalMem = 0;
	hipMemGetInfo(&freeMem, &totalMem);
	printf("GPU Memory avaliable: Free: %lu, Total: %lu\n",freeMem/1024/1024, totalMem/1024/1024);

	unsigned int eachSeqMem = sizeof(char)*max_length_per_seq*2
					+ sizeof(int)*(max_length_per_seq+1)*(max_length_per_seq+1)
					+ sizeof(unsigned int)*3;
	unsigned int batch_size = freeMem * 0.75 / eachSeqMem; // Safety reasons...
	
	hipStream_t * stream1 = new hipStream_t;
	hipStream_t * stream2 = new hipStream_t;

	unsigned int half_b, other_half_b;
	#ifdef DUAL_BUFFERING
	half_b = batch_size / 2;
	other_half_b = batch_size - half_b;
    hipStreamCreate(stream1);
    hipStreamCreate(stream2);
	#else
	half_b = batch_size;
	#endif

	printf("Each batch will be doing this many pairs: %d\n", batch_size);

	////////////////////////////////////////////////////////////////////////////

	// This implementation comes with the free assumption that 
	// all sequences will be having the same size :'(

	char *d_sequence_set1_h1, *d_sequence_set2_h1, *d_sequence_set1_h2, *d_sequence_set2_h2;
	unsigned int *d_pos1_h1, *d_pos2_h1, *d_pos_matrix_h1, *d_pos1_h2, *d_pos2_h2, *d_pos_matrix_h2;
	int *d_score_matrix_h1, *d_score_matrix_h2;

	start_marker = gettime();
	// Allocating memory for both halves

	// First half
	hipMalloc( (void**)&d_sequence_set1_h1, sizeof(char)*(pos1[1]*half_b) );
    hipMalloc( (void**)&d_sequence_set2_h1, sizeof(char)*(pos1[1]*half_b)) ;
    hipMalloc( (void**)&d_score_matrix_h1, sizeof(int)*(pos_matrix[1]*half_b)) ;
    hipMalloc( (void**)&d_pos1_h1, sizeof(unsigned int)*(half_b+1) ) ;
    hipMalloc( (void**)&d_pos2_h1, sizeof(unsigned int)*(half_b+1) ) ;
    hipMalloc( (void**)&d_pos_matrix_h1, sizeof(unsigned int)*(half_b+1) ) ;

    #ifdef DUAL_BUFFERING
    // Second half
    hipMalloc( (void**)&d_sequence_set1_h2, sizeof(char)*(pos1[1]*other_half_b) );
    hipMalloc( (void**)&d_sequence_set2_h2, sizeof(char)*(pos2[1]*other_half_b)) ;
    hipMalloc( (void**)&d_score_matrix_h2, sizeof(int)*(pos_matrix[1]*other_half_b)) ;
    hipMalloc( (void**)&d_pos1_h2, sizeof(unsigned int)*(other_half_b+1) );
    hipMalloc( (void**)&d_pos2_h2, sizeof(unsigned int)*(other_half_b+1) ) ;
    hipMalloc( (void**)&d_pos_matrix_h2, sizeof(unsigned int)*(other_half_b+1) ) ;
	#endif


	fprintf(stdout,"hipMalloc = %f\n", gettime()-start_marker);

	////////////////////////////////////////////////////////////////////////////
	// WARNING BOILERPLATE CODE !
	// Jesus, why I'm doing this? - Huan.

	struct needle_context * internal_ctx = new needle_context;
	internal_ctx->gpu_num = gpu_num;
	internal_ctx->sequence_set1 = sequence_set1;
	internal_ctx->pos1 = pos1;
	internal_ctx->pos2 = pos2;
	internal_ctx->score_matrix = score_matrix;
	internal_ctx->pos_matrix = pos_matrix;
	internal_ctx->d_sequence_set1_h1 = d_sequence_set1_h1;
	internal_ctx->d_sequence_set2_h1 = d_sequence_set2_h1;
	internal_ctx->d_sequence_set1_h2 = d_sequence_set1_h2;
	internal_ctx->d_sequence_set2_h2 = d_sequence_set2_h2;
	internal_ctx->d_pos1_h1 = d_pos1_h1;
	internal_ctx->d_pos2_h1 = d_pos2_h1;
	internal_ctx->d_pos_matrix_h1 = d_pos_matrix_h1;
	internal_ctx->d_pos1_h2 = d_pos1_h2;
	internal_ctx->d_pos2_h2 = d_pos2_h2;
	internal_ctx->d_pos_matrix_h2 = d_pos_matrix_h2;
	internal_ctx->d_score_matrix_h1 = d_score_matrix_h1;
	internal_ctx->d_score_matrix_h2 = d_score_matrix_h2;
	internal_ctx->penalty = -10;
	internal_ctx->stream1 = stream1;
	internal_ctx->stream2 = stream2;
	internal_ctx->half_b = half_b;
	internal_ctx->other_half_b = other_half_b;
	
	printf("-- NEEDLEMAN MODULE INITIALIZING DONE --\n", 0);
	
	return (void *) internal_ctx;
}

void needle_align(void * needle_ctx, int num_pairs) {
	////////////////////////////////////////////////////////////////////////////
	// WARNING BOILERPLATE CODE !

	struct needle_context *internal_ctx = static_cast<struct needle_context *>(needle_ctx);

	needleman_gpu(
		internal_ctx->sequence_set1,
		internal_ctx->sequence_set2,
		internal_ctx->pos1,
		internal_ctx->pos2,
		internal_ctx->score_matrix,
		internal_ctx->pos_matrix,
		num_pairs,
		internal_ctx->penalty,
		internal_ctx->d_sequence_set1_h1,
		internal_ctx->d_sequence_set2_h1,
		internal_ctx->d_sequence_set1_h2,
		internal_ctx->d_sequence_set2_h2,
		internal_ctx->d_pos1_h1,
		internal_ctx->d_pos2_h1,
		internal_ctx->d_pos_matrix_h1,
		internal_ctx->d_pos1_h2,
		internal_ctx->d_pos2_h2,
		internal_ctx->d_pos_matrix_h2,
		internal_ctx->d_score_matrix_h1,
		internal_ctx->d_score_matrix_h2,
		internal_ctx->stream1,
		internal_ctx->stream2,
		internal_ctx
	);
}


void needle_finalize(void * needle_ctx)
{
	struct needle_context *internal_ctx = static_cast<struct needle_context *>(needle_ctx);

	hipFree(internal_ctx->d_sequence_set1_h1);
	hipFree(internal_ctx->d_sequence_set2_h1);
	hipFree(internal_ctx->d_pos1_h1);
	hipFree(internal_ctx->d_pos2_h2);
	hipFree(internal_ctx->d_pos_matrix_h1);
	hipFree(internal_ctx->d_score_matrix_h1);

	#ifdef DUAL_BUFFERING
	hipFree(internal_ctx->d_sequence_set1_h2);
	hipFree(internal_ctx->d_sequence_set2_h2);
	hipFree(internal_ctx->d_pos1_h2);
	hipFree(internal_ctx->d_pos2_h2);
	hipFree(internal_ctx->d_pos_matrix_h2);
	hipFree(internal_ctx->d_score_matrix_h2);

	hipStreamDestroy(*(internal_ctx->stream1));
	hipStreamDestroy(*(internal_ctx->stream2));
	#endif

	delete internal_ctx->stream1;
	delete internal_ctx->stream2;

	delete(internal_ctx);
}
