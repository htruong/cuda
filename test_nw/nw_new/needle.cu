#define LIMIT -999
#define BLOCK_SIZE 16
#define MAX_SEQ_LEN 2096
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <needle.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

// includes, kernels
#include <needle_kernel.cu>

inline void cudaCheckError(int line, hipError_t ce)
{
	if (ce != hipSuccess){
		printf("Error: line %d %s\n", line, hipGetErrorString(ce));
		exit(1);
	}
}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

double gettime(){
  struct timeval t;
  gettimeofday(&t,NULL);
  return t.tv_sec+t.tv_usec*1e-6;
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	runTest( argc, argv);
    return EXIT_SUCCESS;
}

void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <pair number> <penalty> \n", argv[0]);
	fprintf(stderr, "\t<pair number>  - times of comparison\n");
	fprintf(stderr, "\t<penalty> - penalty(negative integer)\n");
	exit(1);
}

void runTest( int argc, char** argv) 
{
	double time, end_time;
	int pair_num;
	short penalty;
    char sequence_set1[MAX_SEQ_LEN*32] = {0}, sequence_set2[MAX_SEQ_LEN*32] = {0};
	unsigned int pos1[33] = {0}, pos2[33] = {0}, pos_matrix[33] = {0};
	short *score_matrix;
	char *d_sequence_set1, *d_sequence_set2;
	unsigned int *d_pos1, *d_pos2, *d_pos_matrix;
	short *d_score_matrix;
	int seq1_len, seq2_len;
    // the lengths of the two sequences should be able to divided by 16.
	// And at current stage  max_rows needs to equal max_cols
	if (argc == 3)
	{
		pair_num = atoi(argv[1]);
		penalty = atoi(argv[2]);
		if (pair_num>32){
			fprintf(stderr, "\t<pair number>  - times of comparison should be less than 32\n");
			exit(1);
		}
	}
    else{
		usage(argc, argv);
    }
	// first API
	time = gettime();
	cudaCheckError( __LINE__, hipSetDevice(0) );
	
	end_time = gettime();
	fprintf(stdout,"First API,%lf\n",end_time-time);
	time = end_time;
	
    srand ( 7 );
	pos_matrix[0] = pos1[0] = pos2[0] = 0;
	for (int i=0; i<pair_num; ++i){
		//please define your own sequence 1.
		seq1_len = 2048+rand() % 20;
		//printf("Seq1 length: %d\n", seq1_len);	
		for (int j=0; j<seq1_len; ++j)		
			sequence_set1[ pos1[i] + j ] = rand() % 20 + 1;
		pos1[i+1] = pos1[i] + seq1_len;
		//please define your own sequence 2.
		seq2_len = 2048+rand() % 20;		
		//printf("Seq2 length: %d\n\n", seq2_len);		
		for (int j=0; j<seq2_len; ++j)		
			sequence_set2[ pos2[i] +j ] = rand() % 20 + 1;
		pos2[i+1] = pos2[i] + seq2_len;
		//printf("Matrix size increase: %d\n", (seq1_len+1) * (seq2_len+1));
		pos_matrix[i+1] = pos_matrix[i] + (seq1_len+1) * (seq2_len+1);
	}
	/*for (int i=0; i<=pair_num; ++i)
		printf("Size of %dth score matrix: %d\n", i, pos_matrix[i]);
	*/score_matrix = (short *)malloc( pos_matrix[pair_num]*sizeof(short));
	
	// printf("Start Needleman-Wunsch\n");
	cudaCheckError( __LINE__, hipMalloc( (void**)&d_sequence_set1, sizeof(char)*pos1[pair_num] ) );
	cudaCheckError( __LINE__, hipMalloc( (void**)&d_sequence_set2, sizeof(char)*pos2[pair_num] ) );
	cudaCheckError( __LINE__, hipMalloc( (void**)&d_score_matrix, sizeof(short)*pos_matrix[pair_num]) );
	cudaCheckError( __LINE__, hipMalloc( (void**)&d_pos1, sizeof(unsigned int)*(pair_num+1) ) );
	cudaCheckError( __LINE__, hipMalloc( (void**)&d_pos2, sizeof(unsigned int)*(pair_num+1) ) );
 	cudaCheckError( __LINE__, hipMalloc( (void**)&d_pos_matrix, sizeof(unsigned int)*(pair_num+1) ) );
	//cudaCheckError( __LINE__, hipMemset( (void**)&d_score_matrix, 0, sizeof(short)*pos_matrix[pair_num]) );
	// CPU phases
	end_time = gettime();
	fprintf(stdout,"CPU,%lf\n",end_time-time);
	time = end_time;
	
	// Memcpy to device
	cudaCheckError( __LINE__, hipMemcpy( d_sequence_set1, sequence_set1, sizeof(char)*pos1[pair_num], hipMemcpyHostToDevice ) );
	cudaCheckError( __LINE__, hipMemcpy( d_sequence_set2, sequence_set2, sizeof(char)*pos2[pair_num], hipMemcpyHostToDevice ) );
	cudaCheckError( __LINE__, hipMemcpy( d_pos1, pos1, sizeof(unsigned int)*(pair_num+1), hipMemcpyHostToDevice ) );
	cudaCheckError( __LINE__, hipMemcpy( d_pos2, pos2, sizeof(unsigned int)*(pair_num+1), hipMemcpyHostToDevice ) );
 	cudaCheckError( __LINE__, hipMemcpy( d_pos_matrix, pos_matrix, sizeof(unsigned int)*(pair_num+1), hipMemcpyHostToDevice ) );
	
	end_time = gettime();
	fprintf(stdout,"Memcpy to device,%lf\n",end_time-time);
	time = end_time;

	//printf("Processing top-left matrix\n");
	//process top-left matrix
	//helloCUDA<<<1, 5>>>(1.2345f);
	//hipDeviceSynchronize();
	/*needleman_cuda_dynamic<<<1, 1024>>>(d_sequence_set1, d_sequence_set2, 
									   d_pos1, d_pos2,
									   d_score_matrix, d_pos_matrix,
									   pair_num, penalty);
	*/
	needleman_cuda_diagonal_global<<<pair_num,32>>>(d_sequence_set1, d_sequence_set2, 
									   d_pos1, d_pos2,
									   d_score_matrix, d_pos_matrix,
									   pair_num, penalty);
	cudaCheckError( __LINE__, hipDeviceSynchronize() );
	end_time = gettime();
	fprintf(stdout,"kernel,%lf\n",end_time-time);
	time = end_time;
	//hipMemset( (void**)&d_score_matrix, 0, sizeof(short)*pos_matrix[pair_num]);
	// Memcpy to host
	cudaCheckError( __LINE__, hipMemcpy( score_matrix, d_score_matrix, sizeof(short)*pos_matrix[pair_num], hipMemcpyDeviceToHost ) );    

	end_time = gettime();
	fprintf(stdout,"Memcpy to host,%lf\n",end_time-time);
	time = end_time;
	

#ifdef TRACEBACK
	for (int i = max_rows - 2,  j = max_rows - 2; i>=0, j>=0;){
		int nw, n, w, traceback;
		if ( i == max_rows - 2 && j == max_rows - 2 )
			//fprintf(fpo, "%d ", output_itemsets[ i * max_cols + j]); //print the first element
		if ( i == 0 && j == 0 )
           break;
		if ( i > 0 && j > 0 ){
			nw = output_itemsets[(i - 1) * max_cols + j - 1];
		    w  = output_itemsets[ i * max_cols + j - 1 ];
            n  = output_itemsets[(i - 1) * max_cols + j];
		}
		else if ( i == 0 ){
		    nw = n = LIMIT;
		    w  = output_itemsets[ i * max_cols + j - 1 ];
		}
		else if ( j == 0 ){
		    nw = w = LIMIT;
            n  = output_itemsets[(i - 1) * max_cols + j];
		}
		else{
		}

		//traceback = maximum(nw, w, n);
		int new_nw, new_w, new_n;
		new_nw = nw + referrence[i * max_cols + j];
		new_w = w - penalty;
		new_n = n - penalty;
		
		traceback = maximum(new_nw, new_w, new_n);
		if(traceback == new_nw)
			traceback = nw;
		if(traceback == new_w)
			traceback = w;
		if(traceback == new_n)
            traceback = n;
			
		//fprintf(fpo, "%d ", traceback);

		if(traceback == nw )
		{i--; j--; continue;}

        else if(traceback == w )
		{j--; continue;}

        else if(traceback == n )
		{i--; continue;};
	}
#endif
//	fclose(fpo);
	hipFree(d_sequence_set1);
	hipFree(d_sequence_set2);
	hipFree(d_pos1);
	hipFree(d_pos2);
 	hipFree(d_pos_matrix);
	hipFree(d_score_matrix);
	free(score_matrix);
	
}
