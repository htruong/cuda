#define LIMIT -999
#define BLOCK_SIZE 16
#define MAX_SEQ_LEN 2100
#define MAX_SEQ_NUM 1024
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "needle.h"
#include "needle_cpu.h"


#define LENGTH 2000
#define TRACEBACK
// includes, kernels
#include "needle_cpu.c"
//#include "needle_kernel_dynamic.cu"
#include "needle_kernel_diagonal.cu"

inline void cudaCheckError(int line, hipError_t ce)
{
    if (ce != hipSuccess) {
        printf("Error: line %d %s\n", line, hipGetErrorString(ce));
        exit(1);
    }
}

int validation(int *score_matrix_cpu, int *score_matrix, unsigned int length)
{
    unsigned int i = 0;
    while (i!=length) {
        if ( (score_matrix_cpu[i]) == (score_matrix[i] >> 2) ) {
            ++i;
            continue;
        }
        else {
            printf("i = %d, expected %d, got %d.\n",i, score_matrix_cpu[i], score_matrix[i] >> 2);
            return 0;
        }
    }
    return 1;
}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

double gettime() {
    struct timeval t;
    gettimeofday(&t,NULL);
    return t.tv_sec+t.tv_usec*1e-6;
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}

void usage(int argc, char **argv)
{
    fprintf(stderr, "Usage: %s <pair number> <penalty> \n", argv[0]);
    fprintf(stderr, "\t<pair number>  - times of comparison\n");
    fprintf(stderr, "\t<penalty> - penalty(negative integer)\n");
    exit(1);
}

void runTest( int argc, char** argv)
{
    double time, end_time;
    int pair_num;
    short penalty;
    char sequence_set1[MAX_SEQ_LEN*MAX_SEQ_NUM] = {0}, sequence_set2[MAX_SEQ_LEN*MAX_SEQ_NUM] = {0};
    unsigned int pos1[MAX_SEQ_NUM] = {0}, pos2[MAX_SEQ_NUM] = {0}, pos_matrix[MAX_SEQ_NUM] = {0};
    int *score_matrix;
    int *trace_matrix;
    int *score_matrix_cpu;
    int *trace_matrix_cpu;
    char *d_sequence_set1, *d_sequence_set2;
    unsigned int *d_pos1, *d_pos2, *d_pos_matrix;
    int *d_score_matrix;
    int seq1_len, seq2_len;

    if (argc == 3)
    {
        pair_num = atoi(argv[1]);
        penalty = atoi(argv[2]);
        if (pair_num>MAX_SEQ_NUM) {
            fprintf(stderr, "\t<number of pairs>  - number of pairs, must be less than %d\n",MAX_SEQ_NUM);
            exit(1);
        }
    }
    else {
        usage(argc, argv);
    }

    // first API
    time = gettime();
    cudaCheckError( __LINE__, hipSetDevice(0) );

    end_time = gettime();
    fprintf(stdout,"First API,%lf\n",end_time-time);
    time = end_time;

    // Get input data

    srand ( 7 );
    pos_matrix[0] = pos1[0] = pos2[0] = 0;
    for (int i=0; i<pair_num; ++i) {
        //please define your own sequence 1
        seq1_len = LENGTH; //64+rand() % 20;
        //printf("Seq1 length: %d\n", seq1_len);
        for (int j=0; j<seq1_len; ++j)
            sequence_set1[ pos1[i] + j ] = rand() % 20 + 'A';
        pos1[i+1] = pos1[i] + seq1_len;
        //please define your own sequence 2.
        seq2_len = LENGTH;//64+rand() % 20;
        //printf("Seq2 length: %d\n\n", seq2_len);
        for (int j=0; j<seq2_len; ++j)
            sequence_set2[ pos2[i] +j ] = rand() % 20 + 'A';
        pos2[i+1] = pos2[i] + seq2_len;
        //printf("Matrix size increase: %d\n", (seq1_len+1) * (seq2_len+1));
        pos_matrix[i+1] = pos_matrix[i] + (seq1_len+1) * (seq2_len+1);
    }
    score_matrix = (int *)malloc( pos_matrix[pair_num]*sizeof(int));
    
    score_matrix_cpu = (int *)malloc( pos_matrix[pair_num]*sizeof(int));	
    
	#ifdef _LP64
	printf ("Running on a 64-bit platform!\n");
	#else
	#endif

	/*
	short M = -1;
	printf("M: "BYTETOBINARYPATTERN" "BYTETOBINARYPATTERN"\n",
		BYTETOBINARY(M>>8), BYTETOBINARY(M));
    */
    
    printf ("Allocating %dMB of memory... \
		(sizeof int=%d bytes, sizeof short=%d bytes)\n",
		pos_matrix[pair_num]*sizeof(int)/1024/1024,
		sizeof(int),
		sizeof(short)
	);
	
    needleman_cpu(sequence_set1, sequence_set2, pos1, pos2, score_matrix_cpu, pos_matrix, pair_num, penalty);

    // printf("Start Needleman-Wunsch\n");

    cudaCheckError( __LINE__, hipMalloc( (void**)&d_sequence_set1, sizeof(char)*pos1[pair_num] ) );
    cudaCheckError( __LINE__, hipMalloc( (void**)&d_sequence_set2, sizeof(char)*pos2[pair_num] ) );
    cudaCheckError( __LINE__, hipMalloc( (void**)&d_score_matrix, sizeof(int)*pos_matrix[pair_num]) );
    cudaCheckError( __LINE__, hipMalloc( (void**)&d_pos1, sizeof(unsigned int)*(pair_num+1) ) );
    cudaCheckError( __LINE__, hipMalloc( (void**)&d_pos2, sizeof(unsigned int)*(pair_num+1) ) );
    cudaCheckError( __LINE__, hipMalloc( (void**)&d_pos_matrix, sizeof(unsigned int)*(pair_num+1) ) );

    // CPU phases
    end_time = gettime();
    fprintf(stdout,"CPU,%lf\n",end_time-time);
    time = end_time;

    // Memcpy to device
    cudaCheckError( __LINE__,
		hipMemcpy( d_sequence_set1, sequence_set1, sizeof(char)*pos1[pair_num], hipMemcpyHostToDevice )
	);
	
    cudaCheckError( __LINE__,
		hipMemcpy( d_sequence_set2, sequence_set2, sizeof(char)*pos2[pair_num], hipMemcpyHostToDevice )
	);
	
    cudaCheckError( __LINE__,
		hipMemcpy( d_pos1, pos1, sizeof(unsigned int)*(pair_num+1), hipMemcpyHostToDevice )
	);
	
    cudaCheckError( __LINE__,
		hipMemcpy( d_pos2, pos2, sizeof(unsigned int)*(pair_num+1), hipMemcpyHostToDevice )
	);
	
    cudaCheckError( __LINE__,
		hipMemcpy( d_pos_matrix, pos_matrix, sizeof(unsigned int)*(pair_num+1), hipMemcpyHostToDevice )
	);

    //end_time = gettime();
    //fprintf(stdout,"Memcpy to device,%lf\n",end_time-time);
    //time = end_time;

    needleman_cuda_diagonal<<<pair_num,512>>>(d_sequence_set1, d_sequence_set2,
            d_pos1, d_pos2,
            d_score_matrix, d_pos_matrix,
            pair_num, penalty);
    cudaCheckError( __LINE__, hipDeviceSynchronize() );
    //end_time = gettime();
    //fprintf(stdout,"kernel,%lf\n",end_time-time);
    //time = end_time;
    // Memcpy to host
    cudaCheckError( __LINE__, hipMemcpy( score_matrix, d_score_matrix, sizeof(int)*pos_matrix[pair_num], hipMemcpyDeviceToHost ) );

    end_time = gettime();
    //fprintf(stdout,"Memcpy to host,%lf\n",end_time-time);
    fprintf(stdout,"Total CUDA implementation time, %lf\n",end_time-time);
    time = end_time;

    if ( validation(score_matrix_cpu, score_matrix, pos_matrix[pair_num]) )
        printf("Validation: PASS\n");
    else
        printf("Validation: FAIL\n");

	#ifdef TRACEBACK
		printf("Here comes the result of the first pair...\n");
		int seq1_begin = pos1[0];
		int seq1_end = pos1[1];
		int seq2_begin = pos2[0];
		int seq2_end = pos2[1];
		int *current_matrix = score_matrix + pos_matrix[0];
		printf("1st seq len = %d =\n%.*s\n", seq1_end - seq1_begin, seq1_end - seq1_begin, sequence_set1 + seq1_begin);
		printf("2nd seq len = %d =\n%.*s\n", seq2_end - seq2_begin, seq2_end - seq2_begin, sequence_set2 + seq2_begin);
		printf("traceback = \n");
		bool done = false;
		int current_pos = ((seq1_end - seq1_begin)+1) * ((seq2_end - seq2_begin)+1) -1; // start at the last cell of the matrix

		for (int i = 0; i < LENGTH + 1; i++) {
			for (int j = 0; j < LENGTH + 1; j++) {
				int dir = current_matrix[i*(LENGTH+1)+j];
				if ((dir & 0x03) == TRACE_UL) {
					printf("\\");
				} else if ((dir & 0x03) == TRACE_U) {
					printf("^");
				} else if ((dir & 0x03) == TRACE_L) {
					printf("<");
				} else {
					printf("-");
				}
			}
			printf("\n");
		}


		for (int i = 0; i < LENGTH + 1; i++) {
			for (int j = 0; j < LENGTH + 1; j++) {
				int dir = current_matrix[i*(LENGTH+1)+j] >> 2;
				printf("%4d ", dir);
			}
			printf("\n");
		}
		
		printf("Actual traceback:\n");
		while (!done) {
			int dir = current_matrix[current_pos];
//			printf("current_pos = %d, dir = %x, score = %d\n", current_pos, dir & 0x03, dir >> 2);
			
			if ((dir & 0x03) == TRACE_UL) {
				printf("\\");
				current_pos = current_pos - (seq1_end - seq1_begin + 1) - 1;
			} else if ((dir & 0x03) == TRACE_U) {
				printf("^");
				current_pos = current_pos - (seq1_end - seq1_begin + 1);
			} else if ((dir & 0x03) == TRACE_L) {
				printf("<");
				current_pos = current_pos - 1;
			} else {
				printf("seems to have reached the origin...");
				done = true;
			}
		}
		printf("traceback done!\n");
	#endif
	
	//	fclose(fpo);
    hipFree(d_sequence_set1);
    hipFree(d_sequence_set2);
    hipFree(d_pos1);
    hipFree(d_pos2);
    hipFree(d_pos_matrix);
    hipFree(d_score_matrix);
    free(score_matrix);

}
