#define LIMIT -999
#define BLOCK_SIZE 16
#define MAX_SEQ_LEN 2100
#define MAX_SEQ_NUM 1024
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

inline void cudaCheckError(int line, hipError_t ce)
{
    if (ce != hipSuccess) {
        printf("Error: line %d %s\n", line, hipGetErrorString(ce));
        exit(1);
    }
}

// HACK Huan's hack
// this is not the updated validation code
int validation(int *score_matrix_cpu, int *score_matrix, unsigned int length)
{
    unsigned int i = 0;
    while (i!=length) {
        if ( (score_matrix_cpu[i]) == (score_matrix[i] >> 2) ) {
            ++i;
            continue;
        }
        else {
            printf("i = %d, expected %d, got %d.\n",i, score_matrix_cpu[i], score_matrix[i] >> 2);
            return 0;
        }
    }
    return 1;
}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

double gettime() {
    struct timeval t;
    gettimeofday(&t,NULL);
    return t.tv_sec+t.tv_usec*1e-6;
}

__global__ void dummy_function(int * array, unsigned int howlarge)
{
		int tid = blockIdx.x*blockDim.x + threadIdx.x;
		for (int i = 0; i < 9; i++) {
			for (int delta=0; delta<howlarge; delta=delta+blockDim.x*gridDim.x) {
				if (tid+delta < howlarge)
		        		array[tid+delta] = array[tid+delta] + tid;	
			}
		}
}


void runTest()
{
    double start, end, now;
    unsigned int nints = 500 * 1024 * 1024;
    unsigned int sz = nints * sizeof(int);

		unsigned int nints_small = 1 * 1024 * 1024;
		unsigned int sz_small = nints_small * sizeof(int);

		#ifdef _LP64
		printf ("Running on a 64-bit platform!\n", 0);
		#else
		#endif
    
    int * dummy_cpu, * dummy_cpu2, * dummy_small_cpu, * dummy_small_cpu2;
    hipHostMalloc( (void**) &dummy_cpu, sz , hipHostMallocDefault);
    hipHostMalloc( (void**) &dummy_cpu2, sz , hipHostMallocDefault);
	hipHostMalloc ( (void**) &dummy_small_cpu, sz_small, hipHostMallocDefault);
	hipHostMalloc ( (void**) &dummy_small_cpu2, sz_small, hipHostMallocDefault);


		int * dummy_gpu, * dummy_gpu2, * dummy_small_gpu, * dummy_small_gpu2;
		hipMalloc( (void**) &dummy_gpu, sz );
		hipMalloc( (void**) &dummy_gpu2, sz );
		hipMalloc( (void**) &dummy_small_gpu, sz_small );
		hipMalloc( (void**) &dummy_small_gpu2, sz_small );

		double kernelt = 0, memcpyt = 0, st = 0, ast = 0;


#define TIMES 5	
    start = gettime();
    dummy_function<<<100,512>>>(dummy_gpu, nints);
    hipDeviceSynchronize();
    end = gettime();
    printf("time for kernel call = %f\n", end-start);
	
    start = gettime();
    hipMemcpy(dummy_cpu, dummy_gpu, sz, hipMemcpyDeviceToHost );
    hipDeviceSynchronize();
    end = gettime();
    printf("time for memcopy D-H = %f\n", end-start);

    start = gettime();
    hipMemcpy(dummy_small_gpu, dummy_small_cpu, sz_small, hipMemcpyHostToDevice);
    dummy_function<<<100,512>>>(dummy_gpu, nints);
    hipMemcpy(dummy_cpu, dummy_gpu, sz, hipMemcpyDeviceToHost );
    hipDeviceSynchronize();
    end = gettime();
    printf("time for one iteration = %f\n", end-start);

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipStream_t stream2;
    hipStreamCreate(&stream2);

#define DEBUG 1
	
    for (int sync=0; sync<2; sync++){

        start = gettime();
	for (int i = 0; i< TIMES; i++) {
	        // small sync copy H->D	
		hipMemcpyAsync(dummy_small_gpu, dummy_small_cpu, sz_small, hipMemcpyHostToDevice, stream1);
                //kernel function
	        dummy_function<<<100,512, 0, stream1>>>(dummy_gpu, nints);
    	        hipDeviceSynchronize();
                //large copy D->H can be sync or async
#ifdef DEBUG		
                now = gettime();
#endif
		if (sync){
		    hipMemcpy(dummy_cpu, dummy_gpu, sz, hipMemcpyDeviceToHost);
                }else{
		    hipMemcpyAsync(dummy_cpu, dummy_gpu, sz, hipMemcpyDeviceToHost, stream1 );
                }
#ifdef DEBUG		
		printf("(A)sync call took %f\n", gettime() - now);
#endif
		// small sync copy H->D 
                hipMemcpyAsync(dummy_small_gpu2, dummy_small_cpu2, sz_small, hipMemcpyHostToDevice, stream2);
                //kernel function
                dummy_function<<<100,512, 0, stream2>>>(dummy_gpu2, nints);
                hipDeviceSynchronize();
                //large copy D->H can be sync or async
                if (sync){
                    hipMemcpy(dummy_cpu2, dummy_gpu2, sz, hipMemcpyDeviceToHost);
                }else{
                    hipMemcpyAsync(dummy_cpu2, dummy_gpu2, sz, hipMemcpyDeviceToHost , stream2);
                }

	}
    	hipDeviceSynchronize();
        end = gettime();
	if (!sync)
		printf("%d iterations: time for ASYNC calls = %f\n",TIMES,end-start);
	else
		printf("%d iterations: time for SYNC calls = %f\n",TIMES,end-start);

   }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
    runTest();
    return EXIT_SUCCESS;
}

